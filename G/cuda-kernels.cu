#include "hip/hip_runtime.h"
__global__ void copyImgKernel(unsigned char* img_in, unsigned char* img_out, int width, int height)
{
   int i = threadIdx.x+blockIdx.x*blockDim.x;
   int j = threadIdx.y+blockIdx.y*blockDim.y;

   if (i<width && j<height)
   {
      int adrIn=(i+j*width)*4;
      int adrOut=adrIn;
      unsigned char r,g,b,a;

      r = img_in[adrIn+0];
      g = img_in[adrIn+1];
      b = img_in[adrIn+2];
      a = img_in[adrIn+3];
    
      img_out[adrOut+0] = r; 
      img_out[adrOut+1] = g;
      img_out[adrOut+2] = b;
      img_out[adrOut+3] = a;
   }
}

__global__ void linearTransformKernel(unsigned char* img_in, unsigned char* img_out, int width, int height, float alpha, float beta)
{
    int i = threadIdx.x+blockIdx.x*blockDim.x;
    int j = threadIdx.y+blockIdx.y*blockDim.y;

    
    if (i<width && j<height)
    {
        int adrIn=(i+j*width)*4;
        int adrOut=adrIn;
        unsigned char r,g,b,a;
        float r_new, g_new, b_new;
        
        r = img_in[adrIn+0];
        g = img_in[adrIn+1];
        b = img_in[adrIn+2];
        a = img_in[adrIn+3];
        
        r_new = alpha*r + beta;             //%\label{line:linearStart}%
        r_new = r_new < 0?     0 : r_new;
        r_new = r_new > 255? 255 : r_new;

        g_new = alpha*g + beta;
        g_new = g_new < 0?     0 : g_new;
        g_new = g_new > 255? 255 : g_new;

        b_new = alpha*b + beta;
        b_new = b_new < 0?     0 : b_new;
        b_new = b_new > 255? 255 : b_new;   //%\label{line:linearEnd}%

        img_out[adrOut+0] = (unsigned char)r_new; 
        img_out[adrOut+1] = (unsigned char)g_new;
        img_out[adrOut+2] = (unsigned char)b_new;
        img_out[adrOut+3] = a;
    }
}

__global__ void mirrorKernel(unsigned char* img_in, unsigned char* img_out, int width, int height)
{
    int i = threadIdx.x+blockIdx.x*blockDim.x;
    int j = threadIdx.y+blockIdx.y*blockDim.y;

    
    if (i<width && j<height)
    {
        int adrIn;
        int adrOut=(i+j*width)*4;
        unsigned char r,g,b,a;
        
        if (i < width/2) {      // //%Adressberechnung nach Formel (\ref{eq:mirror})%
            adrIn=adrOut;
        
        } else {
            adrIn=(width-i+j*width)*4; %\label{line:mirror}%
        }
        r = img_in[adrIn+0];
        g = img_in[adrIn+1];
        b = img_in[adrIn+2];
        a = img_in[adrIn+3];

        img_out[adrOut+0] = r; 
        img_out[adrOut+1] = g;
        img_out[adrOut+2] = b;
        img_out[adrOut+3] = a;
    }
}

__global__ void bwKernel(unsigned char* img_in, unsigned char* img_out, int width, int height)
{
    int i = threadIdx.x+blockIdx.x*blockDim.x;
    int j = threadIdx.y+blockIdx.y*blockDim.y;

    if (i<width && j<height)
    {
        int adrIn=(i+j*width)*4;
        int adrOut=adrIn;
        unsigned char r,g,b,a;
        unsigned char bw;

        r = img_in[adrIn+0];
        g = img_in[adrIn+1];
        b = img_in[adrIn+2];
        a = img_in[adrIn+3];
    
        bw = (r+g+b)/3;

        img_out[adrOut+0] = bw; 
        img_out[adrOut+1] = bw;
        img_out[adrOut+2] = bw;
        img_out[adrOut+3] = a;
    }
}

__global__ void sobelKernel(unsigned char* img_in, unsigned char* img_out, int width, int height)
{
    const float SX[3][3]={{-1,0,1},{-2,0,2},{-1,0,1}};
    const float SY[3][3]={{-1,-2,-1},{0,0,0},{1,2,1}};

    int i = threadIdx.x+blockIdx.x*blockDim.x;
    int j = threadIdx.y+blockIdx.y*blockDim.y;

    if (i<width && j<height)
    {
    	int adrIn;
    	int adrOut=(i+j*width)*4;
        int i_new, j_new, bw;
        unsigned char r,a;

        float gX = 0.0f, gY = 0.0f;

        for (int k = -1; k <= 1; k++) {
            for (int l = -1; l <= 1; l++) {
                i_new = i+k;
                j_new = j+l;

                if (i_new < 0 || i_new > width || j_new < 0 || j_new > height) { //%\label{line:sobelOldIf}%
                    r = 0;
                } else {
                    adrIn=(i_new+j_new*width)*4;
                    r = img_in[adrIn];
                }

                gX += SX[1+k][1+l] * r;
                gY += SY[1+k][1+l] * r;                
            }
        }
        
        adrIn = adrOut;
        a = img_in[adrIn+3];

        bw = sqrtf (gX*gX + gY*gY);

        img_out[adrOut+0] = bw; 
        img_out[adrOut+1] = bw;
        img_out[adrOut+2] = bw;
        img_out[adrOut+3] = a;
    }
}
