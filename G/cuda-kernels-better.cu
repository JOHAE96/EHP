
#include <hip/hip_runtime.h>
__global__ void sobelKernel(unsigned char* img_in, unsigned char* img_out, int width, int height)
{
    const float SX[3][3]={{-1,0,1},{-2,0,2},{-1,0,1}};
    const float SY[3][3]={{-1,-2,-1},{0,0,0},{1,2,1}};

    int i = threadIdx.x+blockIdx.x*blockDim.x;
    int j = threadIdx.y+blockIdx.y*blockDim.y;

    if (i<width && j<height)
    {
    	int adrIn;
    	int adrOut=(i+j*width)*4;
        int i_new, j_new;
        unsigned char r,a;
        float bw, gX = 0.0f, gY = 0.0f;
        
        if (i == 0 || i == width-1 || j == 0 || j == height -1) { //%\label{line:sobelBoarder}%
            bw = 0;
        } else {
            for (int k = -1; k <= 1; k++) {        
                for (int l = -1; l <= 1; l++) {
                    i_new = i+k;
                    j_new = j+l;
    
                    adrIn=(i_new+j_new*width)*4;
                    r = img_in[adrIn];

                    gX += SX[1+k][1+l] * r;         //%\label{line:sobelSum}%
                    gY += SY[1+k][1+l] * r;                
                }
            }                                       

            bw = sqrtf (gX*gX + gY*gY);             //%\label{line:sobelNorm}%
            bw = bw < 0? 0 : bw;
            bw = bw > 255? 255 : bw;
        }
        adrIn = adrOut;
        a = img_in[adrIn+3];        //%\label{line:sobelLoadAlpha}%
        
        img_out[adrOut+0] = (unsigned char)bw; 
        img_out[adrOut+1] = (unsigned char)bw;
        img_out[adrOut+2] = (unsigned char)bw;
        img_out[adrOut+3] = a;
    }
}
