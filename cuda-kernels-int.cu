
#include <hip/hip_runtime.h>
//CUDA Kernels für Verwendung von Integer in Aufgabe 3.2

#define getR(img) (((unsigned int)img&0xFF000000)>>24)
#define getG(img) (((unsigned int)img&0x00FF0000)>>16)
#define getB(img) (((unsigned int)img&0x0000FF00)>>8)
#define getA(img) (((unsigned int)img&0x000000FF))
#define output(r,g,b,a) (((unsigned int)r<<24)+((unsigned int)g<<16)+((unsigned int)b<<8) + ((unsigned int)a<<0))

__global__ void copyImgKernel(unsigned int* img_in, unsigned int* img_out, int width, int height)
{
   int i = threadIdx.x+blockIdx.x*blockDim.x;
   int j = threadIdx.y+blockIdx.y*blockDim.y;

   if (i<width && j<height)
   {
      int adrIn=(i+j*width);
      int adrOut=adrIn;
      unsigned int r,g,b,a,in;
      in = img_in[adrIn];

      r = getR(in);
      g = getG(in);
      b = getB(in);
      a = getA(in);

      img_out[adrOut] = output(r,g,b,a);
   }
}

__global__ void linearTransformKernel(unsigned int* img_in, unsigned int* img_out, int width, int height, float alpha, float beta)
{
  int i = threadIdx.x+blockIdx.x*blockDim.x;
  int j = threadIdx.y+blockIdx.y*blockDim.y;


  if (i<width && j<height)
  {
      int adrIn=(i+j*width);
      int adrOut=adrIn;
      unsigned int r,g,b,a,in;
      float r_new, g_new, b_new;

      in = img_in[adrIn];

      r = getR(in);
      g = getG(in);
      b = getB(in);
      a = getA(in);

      r_new = alpha*r + beta;
      r_new = r_new < 0?     0 : r_new;
      r_new = r_new > 255? 255 : r_new;

      g_new = alpha*g + beta;
      g_new = g_new < 0?     0 : g_new;
      g_new = g_new > 255? 255 : g_new;

      b_new = alpha*b + beta;
      b_new = b_new < 0?     0 : b_new;
      b_new = b_new > 255? 255 : b_new;

      img_out[adrOut] = output((unsigned int)r_new, (unsigned int)g_new, (unsigned int)b_new, a);
  }
}

__global__ void mirrorKernel(unsigned int* img_in, unsigned int* img_out, int width, int height)
{
  int i = threadIdx.x+blockIdx.x*blockDim.x;
  int j = threadIdx.y+blockIdx.y*blockDim.y;


  if (i<width && j<height)
  {
      int adrIn=(i+j*width);
      int adrOut=adrIn;
      unsigned int r,g,b,a,in;

      if (i < width/2) {
          adrIn=adrOut;

      } else {
          adrIn=(width-i+j*width);
      }

      in = img_in[adrIn];

      r = getR(in);
      g = getG(in);
      b = getB(in);
      a = getA(in);

      img_out[adrOut] = output(r,g,b,a);
    }
}

__global__ void bwKernel(unsigned int* img_in, unsigned int* img_out, int width, int height)
{
  int i = threadIdx.x+blockIdx.x*blockDim.x;
  int j = threadIdx.y+blockIdx.y*blockDim.y;

  if (i<width && j<height)
  {
     int adrIn=(i+j*width);
     int adrOut=adrIn;
     unsigned int r,g,b,a,in,bw;
     in = img_in[adrIn];

     r = getR(in);
     g = getG(in);
     b = getB(in);
     a = getA(in);

     bw = (r+b+g)/3;

     img_out[adrOut] = output(bw,bw,bw,a);
  }
}

__global__ void sobelKernel(unsigned int* img_in, unsigned int* img_out, int width, int height)
{

   const float SX[3][3]={{-1,0,1},{-2,0,2},{-1,0,1}};
   const float SY[3][3]={{-1,-2,-1},{0,0,0},{1,2,1}};

   int i = threadIdx.x+blockIdx.x*blockDim.x;
   int j = threadIdx.y+blockIdx.y*blockDim.y;

   if (i<width && j<height)
   {
     int adrIn;
     int adrOut=(i+j*width);
     int i_new, j_new;
     unsigned int r,a,bw;    // is enough since r=g=b in a grayscale picture

     float gX = 0.0f, gY = 0.0f;

     for (int k = -1; k <= 1; k++) {
         for (int l = -1; l <= 1; l++) {
             i_new = i+k;
             j_new = j+l;

             if (i_new < 0 || i_new > width-1 || j_new < 0 || j_new > height-1) {
                 r = 0;
             } else {
               adrIn=(i_new+j_new*width);
               r = getR(img_in[adrIn]);
             }

             gX += SX[1+k][1+l] * r;
             gY += SY[1+k][1+l] * r;
         }
     }

     adrIn = adrOut;
     a = getA(img_in[adrIn]);

     bw = (unsigned int)sqrtf (gX*gX + gY*gY);

     img_out[adrOut] = output(bw,bw,bw,a);
   }

}
